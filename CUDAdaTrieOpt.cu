#include "hip/hip_runtime.h"
/********************************************************************
*@version-0.1 
*@author Liyuqian-����ǰ yuqianfly@gmail.com 
*���пƼ���ѧ�����ѧԺ ������ֲ�ʽ����ʵ����
*ע��: �����пƼ���ѧ�����ѧԺ������ֲ�ʽ�����⣬
*�κθ��ˡ��Ŷӡ��о��ṹ����ҵ��λ�Ȳ��ܶԱ��㷨����ר���������׫д
*���㷨�����ġ�
*�κθ��ˡ��Ŷӡ��о��ṹ����ҵ��λ�����ԶԱ��㷨����ʹ�á��޸ġ���չ��������
*ʹ�ñ��㷨������ɵ���ʧ��������ʹ�������и���
* 
* ʹ����ʾ��
*     1�����ʵ���ϸ����û�й���������������������ϵ
*     2������븽���ʵ���Ϣ�������к�ǿ�����ϵ�ԣ��κβ���ȷ�޸Ķ����ܵ���
* �ִ��쳣��
*     3��ʹ�ñ���������У���������ʧ������һ�Ų�����
*     4������������ǰ����Ҫ�㹻�Ķ�ջ�ռ䣬����10240000bytes
*     5���Ż��汾�� ���Ż��汾���ڱ���ʱ��ѡ��һ�����У���֧��ͬʱ����
*     6�����ִַʿ���ѡ���Ӧ��ͬ�����Ŀ¼��
*     7������δ�����������⣬����������ϵ��
*********************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <cutil_inline.h>
#include <cutil.h>
#include <string.h>
#include <locale.h>

#include "loadDocs.h"
#include <hip/hip_runtime.h>
#include "InitDictionary.h"

#define WORD_SIZE  30
#define WWORD_NUM   15

#ifndef __GLOBALVAR__
#define __GLOBALVAR__

texture<unsigned int, 1, hipReadModeElementType> rTBase; 
texture<     int, 1, hipReadModeElementType> rTCheck; 
texture<unsigned char, 1, hipReadModeElementType> rTStatus;
texture<unsigned int, 1, hipReadModeElementType> rTCharsHash;

unsigned char * d_status;
int  * d_check;
unsigned int  * d_base;
unsigned int  * d_charsHash;

#endif
#if __DEVICE_EMULATION__
bool InitCUDA(void){return true;}
#else
bool InitCUDA(void)
{
	int count = 0;
	int i = 0;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return false;
	}
	hipSetDevice(i);

	printf("CUDA initialized.\n");
	return true;
}
#endif

#ifndef __INITCUDADATRIE__
#define __INITCUDADATRIE__

/**initial the global Variable method.*/
void loadAndBlendTrie(int *h_check,
		unsigned int *h_base,		
		unsigned char *h_status,
		unsigned int *h_charsHash,
		unsigned int h_baseSize,
		unsigned int h_charHashSize){ 

    unsigned int baseMemSize=sizeof(unsigned int)*h_baseSize;
	unsigned int checkMemSize=sizeof(int)*h_baseSize;
	unsigned int statusMemSize=sizeof(unsigned char)*h_baseSize;

	//global base
    CUDA_SAFE_CALL(hipMalloc( (void**)&d_base,baseMemSize));
	CUDA_SAFE_CALL(hipMemset(d_base,0,baseMemSize));
	CUDA_SAFE_CALL(hipMemcpy(d_base,h_base,baseMemSize,hipMemcpyHostToDevice));
    
	//global check
    CUDA_SAFE_CALL(hipMalloc( (void**)&d_check,checkMemSize));
	CUDA_SAFE_CALL(hipMemset(d_check,0,checkMemSize));
	CUDA_SAFE_CALL( hipMemcpy(d_check,h_check,checkMemSize,hipMemcpyHostToDevice));

    //global status
    CUDA_SAFE_CALL(hipMalloc( (void**)&d_status,statusMemSize));
	CUDA_SAFE_CALL(hipMemset(d_status,0,statusMemSize));
	CUDA_SAFE_CALL( hipMemcpy(d_status,h_status,statusMemSize,hipMemcpyHostToDevice));

    //global charsHash
	CUDA_SAFE_CALL(hipMalloc( (void**)&d_charsHash,sizeof(unsigned int)*(h_charHashSize)));
    CUDA_SAFE_CALL(hipMemset(d_charsHash,'\0',sizeof(unsigned int)*(h_charHashSize)));		
	CUDA_SAFE_CALL(hipMemcpy(d_charsHash,h_charsHash,sizeof(unsigned int)*(h_charHashSize),hipMemcpyHostToDevice));

    //blend array
	cutilSafeCall(hipBindTexture(0, rTBase, d_base)); 
    cutilSafeCall(hipBindTexture(0, rTCheck, d_check ));
    cutilSafeCall(hipBindTexture(0, rTStatus, d_status)); 
    cutilSafeCall(hipBindTexture(0, rTCharsHash, d_charsHash ));
    
}

//texture blend
bool initCUDAdatrieOpt(){
    
	 int *h_check;
     unsigned int *h_base;	         
     unsigned char *h_status; 
	 unsigned int  *h_charsHash; 

	 InitDictionary initDic;	
     h_base = initDic.base;
	 h_check = initDic.check;
	 h_status = initDic.status;     //��λ����	
	 h_charsHash = initDic.charsHash;
     loadAndBlendTrie(h_check,h_base,h_status,h_charsHash,318608,65535);
	 
	 free(h_check);
     free(h_base);		
	 free(h_status);
     free(h_charsHash);
	 
	 return true;
}

void deleteTrie(){

	hipUnbindTexture(rTBase); 
    hipUnbindTexture(rTCheck);
	hipUnbindTexture(rTStatus); 
    hipUnbindTexture(rTCharsHash);
	
    cutilSafeCall(hipFree(d_base));
    cutilSafeCall(hipFree(d_check));
	cutilSafeCall(hipFree(d_status));
	cutilSafeCall(hipFree(d_charsHash));
}
#endif
/************************************************************************/
/* ���ִַ�ʵ��                                                              */
/************************************************************************/
/**����ȫƥ��ִ�*/
__device__ void tGetAllWords(unsigned short *w_chars,int posFrom,int posTo,unsigned short *output){   
	int outputIndex=0;
	int t=0,i=posFrom,start=posFrom,end=0,charHashCode=0;
	unsigned char stats='0';
 		
    int  baseValue = 0;
    int  checkValue = 0;
	for (; i <posTo; i++) {
     	end++;
		charHashCode = tex1Dfetch(rTCharsHash,w_chars[i]);
        if( charHashCode<1 ) stats='0';
		else{
		    checkValue=baseValue;
		    baseValue = tex1Dfetch(rTBase,checkValue) + charHashCode;
		    if (tex1Dfetch(rTCheck,baseValue) == checkValue || tex1Dfetch(rTCheck,baseValue) == -1)
			    stats= tex1Dfetch(rTStatus,baseValue);
		    else
			    stats='0';
		}

		switch (stats) {
			case '0':	
				i = start;
				start++;				
				end = 0;
				baseValue = 0;
				break;

			case '2':
				for(t=0;t<end;t++){
					output[outputIndex++]=w_chars[t+start];
				}
                output[outputIndex++]=49;
				break;
			case '3':
				for(t=0;t<end;t++){
					output[outputIndex++]=w_chars[t+start];					    
			    }                
				 output[outputIndex++]=49;
				 i = start;
				 start++;
				 end = 0;
				 baseValue = 0;				
				 break;
			}//end of switch		
	}//end of for			
}


/**�������ƥ��ִ�*/
__device__ void tMaxFrontWords(unsigned short * w_chars,int posFrom,int posTo,unsigned short * output) {
	int outputIndex=0;

	int t=0,i=posFrom,start=posFrom,end=0,charHashCode=0;
	unsigned char stats='0';
 		
    int  tempEnd = 0;
    int  baseValue = 0;
    int  checkValue = 0;
    bool hasEnd = false;
    int wlen=posTo-posFrom;
	for(;i<posTo;i++){
    	end++;
		charHashCode = tex1Dfetch(rTCharsHash,w_chars[i]);
		if( charHashCode<1 ) stats='0';
		else{
             checkValue=baseValue;
		     baseValue = tex1Dfetch(rTBase,checkValue) + charHashCode;
		     if (tex1Dfetch(rTCheck,baseValue) == checkValue || tex1Dfetch(rTCheck,baseValue) == -1)
			    stats= tex1Dfetch(rTStatus,baseValue);
		     else
			    stats='0';
		}

		switch (stats) {
		case '0':
			if (hasEnd) {
				for(t=0;t<tempEnd;t++){
					output[outputIndex++]=w_chars[t+start];
				}
				output[outputIndex++]=49;
				hasEnd = false;
				baseValue = 0;
				start = start + tempEnd ;
				i = start-1;
				tempEnd = 0;
				end = 0;				
				break;
			} else {
				baseValue = 0;
				tempEnd = 0;
				i = start;
				start++;				
				end = 0;
			}
			break;
		case '2':
			tempEnd = end;
			hasEnd = true;
			break;
		case '3':
			for(t=0;t<end;t++){
				output[outputIndex++]=w_chars[t+start];			
			}
			output[outputIndex++]=49;//�����ַ�1           		
			hasEnd = false;
			baseValue = 0;
			tempEnd = 0;
			start = i ;
			end = 0;				
			break;
		}
		if (i == wlen - 1) {
			if (hasEnd) {
				for(t=0;t<tempEnd;t++){
					output[outputIndex++]=w_chars[t+start];
				}
				output[outputIndex++]=49;	
				hasEnd = false;
				baseValue = 0;
				start = start + tempEnd;
				i = start-1;
				tempEnd = 0;
				end = 0;
				break;
					
			}
		}
	}
}

/**������Сƥ��ִ�*/
__device__ void tMinFrontWords(unsigned short * w_chars,int posFrom,int posTo,unsigned short * output){
  	
    int outputIndex=0;
	int t=0,i=posFrom,start=posFrom,end=0,charHashCode=0;
	unsigned char stats='0';
    int  baseValue = 0;
    int  checkValue = 0;

	for (; i < posTo; i++) {
		end++;
		charHashCode = tex1Dfetch(rTCharsHash,w_chars[i]);
        if( charHashCode<1 ) stats='0';
		else{
        checkValue=baseValue;
		baseValue = tex1Dfetch(rTBase,checkValue) + charHashCode;
		if (tex1Dfetch(rTCheck,baseValue) == checkValue || tex1Dfetch(rTCheck,baseValue) == -1)
			stats= tex1Dfetch(rTStatus,baseValue);
		else
			stats='0';
		}
		switch (stats) {
			case '0':
				baseValue = 0;
				i = start;
				start++;
				end = 0;
				break;
			case '2':
				for(t=0;t<end;t++)	{
					output[outputIndex++]=w_chars[t+start];
				}
				output[outputIndex++]=49;
				baseValue = 0;
				start = i+1;
				end = 0;
				break;
			case '3':
				for(t=0;t<end;t++){
					output[outputIndex++]=w_chars[t+start];
				}
				output[outputIndex++]=49;
				baseValue = 0;
				start = i+1;
				end = 0;
				break;
			}
		}
}

/**�ں���ں���
* ���ܣ����������ĵ��ִ�
* �ĵ����߳���ƽ�����֣�ÿ���ĵ���Ӧһ��block
*/
__global__  void tBatchSearchKernel(HostDocs * inputDocs,HostDocsTotalTokens *outputTokens){ 	
	int bid=blockIdx.x; //��ȫ��id
	int tid=blockIdx.x*blockDim.x+threadIdx.x;//�߳�ȫ��id
	int docsize=inputDocs->DocStreamSize[bid];//���Ӧ�ĵ���С
	int average=docsize/blockDim.x;//ÿ���߳�����
	int start=threadIdx.x*average;//�����˵�
	int end=start+average;//�������˵�
	//tGetAllWords(inputDocs->DocStream[bid],start,end,outputTokens->ThreadsTokens[tid]);
    //tMaxFrontWords(inputDocs->DocStream[bid],start,end,outputTokens->ThreadsTokens[tid]);
    tMinFrontWords(inputDocs->DocStream[bid],start,end,outputTokens->ThreadsTokens[tid]);
}

/**
����汾���ܣ�
���ݼ��ص��ĵ�������������Ӧ��block����ÿ��block����TREAD_PER_BLOCK�߳�
�ִʽ������thread��λ���棬�� block_num* TREAD_PER_BLOCK ������Ԫ�أ�
ÿ��Ԫ�س���MAX_TOKEN_PER��THREAD==100 ��ÿ���̷ִ߳ʽ�����100��������
*/
void runCUDADATrieOpt(char * inputFold,char * outputFold){  
	// make double trie
	if(initCUDAdatrieOpt())
	   printf("InitCUDADAtrie success.\n\n");
    else
	   printf("*** initCUDADATrie failed!\n\n");
	
	//���ļ���inputFold���������ĵ���������������Ҫ������������DOC_BATCH_SIZE==96
	 HostDocs *hdocs = loadBatchDocs(inputFold);
     printHostDocs("docs",hdocs);

	 printf("\nCopy docs to GPU...\n");
	 HostDocs *ddocs;
	 unsigned short **CPU_ARRAY;
	 CPU_ARRAY =(unsigned short **)malloc(sizeof(unsigned short*)*DOC_BATCH_SIZE);
	 memset(CPU_ARRAY,0,sizeof(unsigned short*)*DOC_BATCH_SIZE);

	 int docSize=0,docsNum=hdocs->DocCount;
	 for(int i=0;i<docsNum;i++){
		 docSize=hdocs->DocStreamSize[i];
         cutilSafeCall( hipMalloc((void **)&CPU_ARRAY[i],sizeof(unsigned short)*docSize));
         cutilSafeCall( hipMemset(CPU_ARRAY[i],0,sizeof(unsigned short)*(docSize)));
         cutilSafeCall( hipMemcpy(CPU_ARRAY[i],hdocs->DocStream[i],sizeof(unsigned short)*docSize,hipMemcpyHostToDevice));
	 }   
	cutilSafeCall(hipMalloc( (void**)&ddocs,sizeof(HostDocs)));
	cutilSafeCall(hipMemcpy(ddocs->DocStream,CPU_ARRAY,sizeof(unsigned short*)*DOC_BATCH_SIZE,hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(ddocs->DocStreamSize,hdocs->DocStreamSize,sizeof(unsigned short)*DOC_BATCH_SIZE,hipMemcpyHostToDevice));
	printf("End of copy\n\n");
     
	//printHostDocs("d_docs test",bdocs); 

	//cpu�˽����ں�������
	HostDocsTotalTokens *hDocAllTokens;
	int tokensTotalMemSize=TOTAL_THREADS_NUM*MAX_TOKEN_PER��THREAD;//128*96*100
    hDocAllTokens=(HostDocsTotalTokens*)malloc(sizeof(HostDocsTotalTokens));
	hDocAllTokens->threadsNum=0;
	memset(hDocAllTokens->ThreadsTokens,0,sizeof(unsigned short)*tokensTotalMemSize);
	 
	 //�ں�������
	HostDocsTotalTokens *dDocAllTokens;
    CUDA_SAFE_CALL(hipMalloc( (void**)&dDocAllTokens,sizeof(HostDocsTotalTokens)));
	int tNum=docsNum*TREAD_PER_BLOCK;//ȫ���߳���Ŀ2*128
	cutilSafeCall(hipMemcpy( &dDocAllTokens->threadsNum,&tNum,sizeof(unsigned short),hipMemcpyHostToDevice));
	cutilSafeCall(hipMemset( dDocAllTokens->ThreadsTokens,0,sizeof(unsigned short)*tokensTotalMemSize));
	
	int blockNum=docsNum;//�����߳̿���Ŀ
	int threadsPerBlock=TREAD_PER_BLOCK;//ÿ���߳̿�������̸߳���
    
	dim3 dimBlock(threadsPerBlock,1,1);
	dim3 dimGrid(blockNum,1);
    printf("start kernel...\n");
    unsigned int timer = 0;
    cutilCheckError( cutCreateTimer( &timer));
    cutilCheckError( cutStartTimer( timer));
	
	tBatchSearchKernel<<<dimGrid,dimBlock>>>(ddocs,dDocAllTokens); 
    //testLoad<<<1,1>>>(ddocs,writeDoc);
	cutilCheckMsg("Kernel execution failed\n");	
	hipDeviceSynchronize();
	
    cutilCheckError( cutStopTimer( timer));
    printf("Kernel processing time: %f (ms)\n", cutGetTimerValue( timer));
    cutilCheckError( cutDeleteTimer( timer));
    printf("end of kernel\n");

	cutilSafeCall(hipMemcpy(hDocAllTokens,dDocAllTokens,sizeof(HostDocsTotalTokens),hipMemcpyDeviceToHost));
	writeDocsTotalTokens("keneal docs total tokens: minWords",outputFold,hDocAllTokens);

	//�ͷ���Դ
    free(hdocs);
    free(hDocAllTokens);
	cutilSafeCall(hipFree(ddocs));
	cutilSafeCall(hipFree(dDocAllTokens));	
   
	deleteTrie();
}


int main(int argc, char* argv[])
{
	if(!InitCUDA()) {
		return 0;
	} 
   
	char *console="outputFiles/OminWords_log_48p_64t.txt";
    //freopen(console,"w",stdout); //����ض���������ݽ�������out.txt�ļ��� 
    //time_t timep;
    //time (&timep);
	//printf("------------------------\n");
    //printf("%s\n",ctime(&timep));
	char * inputFold="inputFiles/48/";
	char * outputFold="outputFiles/OminWords_48p_64t.txt";
    runCUDADATrieOpt(inputFold,outputFold);

	//time (&timep);	
	//printf("%s\n",ctime(&timep));
    //printf("------------------------\n");
    //fclose(stdout);//�ر��ļ� 

	CUT_EXIT(argc, argv);   
	return 0;
}
