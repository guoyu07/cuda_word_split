#include "hip/hip_runtime.h"
/********************************************************************
*@version-0.1 
*@author Liyuqian-����ǰ yuqianfly@gmail.com 
*���пƼ���ѧ�����ѧԺ ������ֲ�ʽ����ʵ����
*ע��: �����пƼ���ѧ�����ѧԺ������ֲ�ʽ�����⣬
*�κθ��ˡ��Ŷӡ��о��ṹ����ҵ��λ�Ȳ��ܶԱ��㷨����ר���������׫д
*���㷨�����ġ�
*�κθ��ˡ��Ŷӡ��о��ṹ����ҵ��λ�����ԶԱ��㷨����ʹ�á��޸ġ���չ��������
*ʹ�ñ��㷨������ɵ���ʧ��������ʹ�������и���
* 
* ʹ����ʾ��
*     1�����ʵ���ϸ����û�й���������������������ϵ
*     2������븽���ʵ���Ϣ�������к�ǿ�����ϵ�ԣ��κβ���ȷ�޸Ķ����ܵ���
* �ִ��쳣��
*     3��ʹ�ñ���������У���������ʧ������һ�Ų�����
*     4������������ǰ����Ҫ�㹻�Ķ�ջ�ռ䣬����10240000bytes
*     5���Ż��汾�� ���Ż��汾���ڱ���ʱ��ѡ��һ�����У���֧��ͬʱ����
*     6�����ִַʿ���ѡ���Ӧ��ͬ�����Ŀ¼��
*     7������δ�����������⣬����������ϵ��
*********************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <cutil_inline.h>
#include <cutil.h>
#include <string.h>
#include <locale.h>

#include "dLoadTrie.h"
#include "loadDocs.h"


#define WORD_SIZE  30
#define WWORD_NUM   15

#ifndef __GLOBALVAR__
#define __GLOBALVAR__

__device__  unsigned  char d_Status[318609];//ȫ�ֱ���
__device__   int          d_Check[318608];  //ȫ�ֱ���
__device__  unsigned int  d_Base[318608];  //ȫ�ֱ���
__device__  unsigned int  d_CharsHash[65535];

#endif
#if __DEVICE_EMULATION__
bool InitCUDA(void){return true;}
#else
bool InitCUDA(void)
{
	int count = 0;
	int i = 0;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return false;
	}
	hipSetDevice(i);

	printf("CUDA initialized.\n");
	return true;
}
#endif

/************************************************************************/
/* ���ִַ�ʵ��                                                              */
/************************************************************************/
/**����ȫƥ��ִ�*/
__device__ void gGetAllWords(unsigned short *w_chars,int posFrom,int posTo,unsigned short *output){   
	int outputIndex=0;
	int t=0,i=posFrom,start=posFrom,end=0,charHashCode=0;
	unsigned char stats='0';
 		
    int  baseValue = 0;
    int  checkValue = 0;
	for (; i <posTo; i++) {
     	end++;
		charHashCode = d_CharsHash[w_chars[i]];
        if( charHashCode<1 ) stats='0';
		else{
		    checkValue=baseValue;
		    baseValue = d_Base[checkValue] + charHashCode;
		    if (d_Check[baseValue] == checkValue || d_Check[baseValue] == -1)
			    stats= d_Status[baseValue];
		    else
			    stats='0';
		}

		switch (stats) {
			case '0':	
				i = start;
				start++;				
				end = 0;
				baseValue = 0;
				break;

			case '2':
				for(t=0;t<end;t++){
					 output[outputIndex++]=w_chars[t+start];
				}
                output[outputIndex++]=49;
				break;
			case '3':
				for(t=0;t<end;t++){
					output[outputIndex++]=w_chars[t+start];	
			    }
				 output[outputIndex++]=49;
				 i = start;
				 start++;
				 end = 0;
				 baseValue = 0;				
				 break;
			}//end of switch		
	}//end of for			
}


/**�������ƥ��ִ�*/
__device__ void gMaxFrontWords(unsigned short * w_chars,int posFrom,int posTo,unsigned short * output) {
	int outputIndex=0;

	int t=0,i=posFrom,start=posFrom,end=0,charHashCode=0;
	unsigned char stats='0';
 		
    int  tempEnd = 0;
    int  baseValue = 0;
    int  checkValue = 0;
    bool hasEnd = false;
    int wlen=posTo-posFrom;
	for(;i<posTo;i++){
    	end++;
		charHashCode = d_CharsHash[w_chars[i]];
		if( charHashCode<1 ) stats='0';
		else{
             checkValue=baseValue;
		     baseValue = d_Base[checkValue] + charHashCode;
		     if (d_Check[baseValue] == checkValue || d_Check[baseValue] == -1)
			    stats= d_Status[baseValue];
		     else
			    stats='0';
		}

		switch (stats) {
		case '0':
			if (hasEnd) {
				for(t=0;t<tempEnd;t++){
					output[outputIndex++]=w_chars[t+start];
				}
				output[outputIndex++]=49;
				hasEnd = false;
				baseValue = 0;
				start = start + tempEnd ;
				i = start-1;
				tempEnd = 0;
				end = 0;				
				break;
			} else {
				baseValue = 0;
				tempEnd = 0;
				i = start;
				start++;				
				end = 0;
			}
			break;
		case '2':
			tempEnd = end;
			hasEnd = true;
			break;
		case '3':
			for(t=0;t<end;t++){
				output[outputIndex++]=w_chars[t+start];			
			}
			output[outputIndex++]=49;//�����ַ�1           		
			hasEnd = false;
			baseValue = 0;
			tempEnd = 0;
			start = i ;
			end = 0;				
			break;
		}
		if (i == wlen - 1) {
			if (hasEnd) {
				for(t=0;t<tempEnd;t++){
					output[outputIndex++]=w_chars[t+start];
				}
				output[outputIndex++]=49;	
				hasEnd = false;
				baseValue = 0;
				start = start + tempEnd;
				i = start-1;
				tempEnd = 0;
				end = 0;
				break;
					
			}
		}
	}
}

/**������Сƥ��ִ�*/
__device__ void gMinFrontWords(unsigned short * w_chars,int posFrom,int posTo,unsigned short * output){
  	
    int outputIndex=0;
	int t=0,i=posFrom,start=posFrom,end=0,charHashCode=0;
	unsigned char stats='0';
 		
    int  baseValue = 0;
    int  checkValue = 0;

	for (; i < posTo; i++) {
		end++;
		charHashCode = d_CharsHash[w_chars[i]];       

        if( charHashCode<1 ) stats='0';
		else{
        checkValue=baseValue;
		baseValue = d_Base[checkValue] + charHashCode;
		if (d_Check[baseValue] == checkValue || d_Check[baseValue] == -1)
			stats= d_Status[baseValue];
		else
			stats='0';
		}
		switch (stats) {
			case '0':
				baseValue = 0;
				i = start;
				start++;
				end = 0;
				break;
			case '2':
				for(t=0;t<end;t++)	{
					output[outputIndex++]=w_chars[t+start];
				}
				output[outputIndex++]=49;
				baseValue = 0;
				start = i+1;
				end = 0;
				break;
			case '3':
				for(t=0;t<end;t++){
					output[outputIndex++]=w_chars[t+start];
				}
				output[outputIndex++]=49;
				baseValue = 0;
				start = i+1;
				end = 0;
				break;
			}
		}
}

/**�ں���ں���
* ���ܣ����������ĵ��ִ�
* �ĵ����߳���ƽ�����֣�ÿ���ĵ���Ӧһ��block
*/
__global__  void gBatchSearchKernel(HostDocs * inputDocs,HostDocsTotalTokens *outputTokens){ 	
	int bid=blockIdx.x; //��ȫ��id
	int tid=blockIdx.x*blockDim.x+threadIdx.x;//�߳�ȫ��id
	int docsize=inputDocs->DocStreamSize[bid];//���Ӧ�ĵ���С
	int average=docsize/blockDim.x;//ÿ���߳�����
	int start=threadIdx.x*average;//�����˵�
	int end=start+average;//�������˵�
	//gGetAllWords(inputDocs->DocStream[bid],start,end,outputTokens->ThreadsTokens[tid]);
    //gMaxFrontWords(inputDocs->DocStream[bid],start,end,outputTokens->ThreadsTokens[tid]);
    gMinFrontWords(inputDocs->DocStream[bid],start,end,outputTokens->ThreadsTokens[tid]);
}

/**test load doc*/
__global__ void testLoad(HostDocs * inputDocs,unsigned short * writeadoc){   
	for(int i=0;i<100000;i++)   
	  writeadoc[i]=inputDocs->DocStream[1][i];
}


/**
����汾���ܣ�
���ݼ��ص��ĵ�������������Ӧ��block����ÿ��block����TREAD_PER_BLOCK�߳�
�ִʽ������thread��λ���棬�� block_num* TREAD_PER_BLOCK ������Ԫ�أ�
ÿ��Ԫ�س���MAX_TOKEN_PER��THREAD==100 ��ÿ���̷ִ߳ʽ�����100��������
*/
void runCUDADATrie(char * inputFold,char * outputFold){  
	// make double trie
	if( h_initCUDADATrie())
	   printf("InitCUDADAtrie success.\n\n");
    else
	   printf("*** initCUDADATrie failed!\n\n");
	
	//���ļ���inputFold���������ĵ���������������Ҫ������������DOC_BATCH_SIZE==96
	 HostDocs *hdocs = loadBatchDocs(inputFold);
     printHostDocs("docs",hdocs);

	 printf("\nCopy docs to GPU...\n");
	 HostDocs *ddocs;
	 unsigned short **CPU_ARRAY;
	 CPU_ARRAY =(unsigned short **)malloc(sizeof(unsigned short*)*DOC_BATCH_SIZE);
	 memset(CPU_ARRAY,0,sizeof(unsigned short*)*DOC_BATCH_SIZE);

	 int docSize=0,docsNum=hdocs->DocCount;
	 for(int i=0;i<docsNum;i++){
		 docSize=hdocs->DocStreamSize[i];
         cutilSafeCall( hipMalloc((void **)&CPU_ARRAY[i],sizeof(unsigned short)*docSize));
         cutilSafeCall( hipMemset(CPU_ARRAY[i],0,sizeof(unsigned short)*(docSize)));
         cutilSafeCall( hipMemcpy(CPU_ARRAY[i],hdocs->DocStream[i],sizeof(unsigned short)*docSize,hipMemcpyHostToDevice));
	 }   
	cutilSafeCall(hipMalloc( (void**)&ddocs,sizeof(HostDocs)));
	cutilSafeCall(hipMemcpy(ddocs->DocStream,CPU_ARRAY,sizeof(unsigned short*)*DOC_BATCH_SIZE,hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(ddocs->DocStreamSize,hdocs->DocStreamSize,sizeof(unsigned short)*DOC_BATCH_SIZE,hipMemcpyHostToDevice));
	printf("End of copy\n\n");
     
	//printHostDocs("d_docs test",bdocs);
    	 
	 //cpu�˽����ں�������
	HostDocsTotalTokens *hDocAllTokens;
	int tokensTotalMemSize=TOTAL_THREADS_NUM*MAX_TOKEN_PER��THREAD;//128*96*100
    hDocAllTokens=(HostDocsTotalTokens*)malloc(sizeof(HostDocsTotalTokens));
	hDocAllTokens->threadsNum=0;
	memset(hDocAllTokens->ThreadsTokens,0,sizeof(unsigned short)*tokensTotalMemSize);
	 
	 //�ں�������
	HostDocsTotalTokens *dDocAllTokens;
    CUDA_SAFE_CALL(hipMalloc( (void**)&dDocAllTokens,sizeof(HostDocsTotalTokens)));
	int tNum=docsNum*TREAD_PER_BLOCK;//ȫ���߳���Ŀ2*128
	cutilSafeCall(hipMemcpy( &dDocAllTokens->threadsNum,&tNum,sizeof(unsigned short),hipMemcpyHostToDevice));
	cutilSafeCall(hipMemset( dDocAllTokens->ThreadsTokens,0,sizeof(unsigned short)*tokensTotalMemSize));
	
	int blockNum=docsNum;//�����߳̿���Ŀ
	int threadsPerBlock=TREAD_PER_BLOCK;//ÿ���߳̿�������̸߳���
    
	dim3 dimBlock(threadsPerBlock,1,1);
	dim3 dimGrid(blockNum,1);
    printf("start kernel...\n");
    unsigned int timer = 0;
    cutilCheckError( cutCreateTimer( &timer));
    cutilCheckError( cutStartTimer( timer));
//�����ں�
/**test load code*/
/*
unsigned short *writeDoc;
size_t docMemSize=sizeof(unsigned short)*MAX_DOC_SIZE;
cutilSafeCall(hipMalloc((void**)&writeDoc,docMemSize));
cutilSafeCall(hipMemset(writeDoc,0,docMemSize));
	
unsigned short *readDoc;
readDoc=(unsigned short*)malloc(docMemSize);
memset(readDoc,0,docMemSize);

printf("init..\n");
for(int i=0;i<10;i++)
    printf("%4d: %wc\n",i,readDoc[i]);
*/

	gBatchSearchKernel<<<dimGrid,dimBlock>>>(ddocs,dDocAllTokens); 
//testLoad<<<1,1>>>(ddocs,writeDoc);
	cutilCheckMsg("Kernel execution failed\n");	
	hipDeviceSynchronize();
	
    cutilCheckError( cutStopTimer( timer));
    printf("Kernel processing time: %f (ms)\n", cutGetTimerValue( timer));
    cutilCheckError( cutDeleteTimer( timer));
    printf("end of kernel\n");

//test load code
/*  
cutilSafeCall(hipMemcpy(readDoc,writeDoc,docMemSize,hipMemcpyDeviceToHost));
printf("the contrent:\n");
for(int i=0;i<10;i++)
   printf("%4d : %wc\n",i,readDoc[i]);
printf("%ws\n",readDoc);
*/

	cutilSafeCall(hipMemcpy(hDocAllTokens,dDocAllTokens,sizeof(HostDocsTotalTokens),hipMemcpyDeviceToHost));
	writeDocsTotalTokens("keneal docs total tokens: minWords",outputFold,hDocAllTokens);

	//�ͷ���Դ
    free(hdocs);
    free(hDocAllTokens);
	cutilSafeCall(hipFree(ddocs));
	cutilSafeCall(hipFree(dDocAllTokens));
    /*
	cutilSafeCall(hipFree(d_Base));
	cutilSafeCall(hipFree(d_Check));
	cutilSafeCall(hipFree(d_Status));
	cutilSafeCall(hipFree(d_CharsHash));
	*/
}

/*
int main(int argc, char* argv[])
{
	if(!InitCUDA()) {
		return 0;
	} 
   
	char *console="outputFiles/minWords_log_48_64.txt";
    freopen(console,"w",stdout); //����ض���������ݽ�������out.txt�ļ��� 
    time_t timep;
    time (&timep);
	printf("------------------------\n");
    printf("%s\n",ctime(&timep));
	char * inputFold="inputFiles/48/";
	char * outputFold="outputFiles/minWords_48_64.txt";
    runCUDADATrie(inputFold,outputFold);

	time (&timep);	
	printf("%s\n",ctime(&timep));
    printf("------------------------\n");
    fclose(stdout);//�ر��ļ� 

	CUT_EXIT(argc, argv);
	
    fclose(stdout);//�ر��ļ� 
	return 0;
}
*/